#include "hip/hip_runtime.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/tensor/resize_impl.h"

namespace onnxruntime {
namespace cuda {
template <typename T>
__global__ void _ResizeNearestKernel(const size_t rank,
                                     const int64_t* input_pitches,
                                     const fast_divmod* output_div_pitches,
                                     const float* scales,
                                     const T* input_data,
                                     T* output_data,
                                     const size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

  int div, mod;
  for (int dim = 0; dim < rank; ++dim) {
    output_div_pitches[dim].divmod(output_index, div, mod);
    output_index = mod;
    if (scales[dim] <= 1) {  //downsample
      div = std::ceil(div / scales[dim]);
    } else {  //upsample
      div = div / scales[dim];
    }
    input_index += input_pitches[dim] * div;
  }
  output_data[id] = input_data[input_index];
}

template <typename T>
__global__ void _ResizeBilinearKernel(const int64_t input_dim2,
                                      const int64_t* input_pitches,
                                      const fast_divmod* output_div_pitches,
                                      const float* scales,
                                      const T* input_data,
                                      T* output_data,
                                      const size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;

  // For bilinear mode, scales[0]=scales[1]=1
  int mod;
  int index_of_dim0, index_of_dim1, index_of_dim2, index_of_dim3;
  output_div_pitches[0].divmod(id, index_of_dim0, mod);
  output_div_pitches[1].divmod(mod, index_of_dim1, mod);
  output_div_pitches[2].divmod(mod, index_of_dim2, mod);
  index_of_dim3 = mod;
  int index_of_input_dim2, index_of_input_dim3;
  float x_offset_0, y_offset_0, x_offset_1, y_offset_1;
  index_of_input_dim2 = static_cast<int64_t>(index_of_dim2 / scales[2]);
  index_of_input_dim3 = static_cast<int64_t>(index_of_dim3 / scales[3]);
  input_index = index_of_dim0 * input_pitches[0] +
                index_of_dim1 * input_pitches[1] +
                index_of_input_dim2 * input_pitches[2] +
                index_of_input_dim3;

  T x00 = input_data[input_index];
  T x10, x01, x11;

  bool end_of_dim2 = false, end_of_dim3 = false;
  if (index_of_input_dim2 == (input_dim2 - 1)) {
    // It's the end in dimension 2
    x01 = x00;
    end_of_dim2 = true;
  } else {
    x01 = input_data[input_index + input_pitches[2]];
  }

  if (index_of_input_dim3 == (input_pitches[2] - 1)) {
    // It's the end in dimension 3
    x10 = x00;
    x11 = x01;
    end_of_dim3 = true;
  } else {
    x10 = input_data[input_index + 1];
    x11 = end_of_dim2 ? x10 : input_data[input_index + input_pitches[2] + 1];
  }

  y_offset_0 = end_of_dim2 ? 0.5f : index_of_dim2 / scales[2] - index_of_input_dim2;
  y_offset_1 = 1.0f - y_offset_0;
  x_offset_0 = end_of_dim3 ? 0.5f : index_of_dim3 / scales[3] - index_of_input_dim3;
  x_offset_1 = 1.0f - x_offset_0;

  output_data[id] =
      x00 * static_cast<T>(y_offset_1 * x_offset_1) +
      x01 * static_cast<T>(y_offset_0 * x_offset_1) +
      x10 * static_cast<T>(y_offset_1 * x_offset_0) +
      x11 * static_cast<T>(y_offset_0 * x_offset_0);
}

template <typename T>
void ResizeImpl(const onnxruntime::UpsampleMode upsample_mode,
                const size_t rank,
                const int64_t input_dim2,
                const int64_t* input_pitches,
                const fast_divmod* output_div_pitches,
                const float* scales_vals,
                const T* input_data,
                T* output_data,
                const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  if (onnxruntime::UpsampleMode::NN == upsample_mode) {
    _ResizeNearestKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        rank, input_pitches, output_div_pitches, scales_vals,
        input_data, output_data, N);
  } else if (onnxruntime::UpsampleMode::LINEAR == upsample_mode) {
    _ResizeBilinearKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        input_dim2, input_pitches, output_div_pitches, scales_vals,
        input_data, output_data, N);
  }
}

#define SPECIALIZED_IMPL(T)                                                  \
  template void ResizeImpl<T>(const onnxruntime::UpsampleMode upsample_mode, \
                              const size_t rank,                             \
                              const int64_t input_dim2,                      \
                              const int64_t* input_pitches,                  \
                              const fast_divmod* output_div_pitches,         \
                              const float* scales_vals,                      \
                              const T* input_data,                           \
                              T* output_data,                                \
                              const size_t N);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(half)
SPECIALIZED_IMPL(int32_t)
SPECIALIZED_IMPL(uint8_t)

}  // namespace cuda
}  // namespace onnxruntime
