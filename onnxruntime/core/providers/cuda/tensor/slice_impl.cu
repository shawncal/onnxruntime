#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "slice_impl.h"
#include "core/providers/cuda/cuda_common.h"

namespace onnxruntime {
namespace cuda {

template <typename T >
__global__ void _SliceKernel(const int32_t dimension_count,
                             const int64_t* starts,
                             const int64_t* steps,
                             const int64_t* input_strides,
                             const fast_divmod* div_strides,
                             const T* input_data,
                             T* output_data,
                             const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  int div;
  int mod = id;
  int value = id;
  int dim_idx = 0;
  for (; dim_idx < dimension_count - 1; ++dim_idx) {
    div_strides[dim_idx].divmod(value, div, mod);
    input_index += (starts[dim_idx] + div * steps[dim_idx]) * input_strides[dim_idx];
    value = mod;
  }
  input_index += starts[dim_idx] + mod * steps[dim_idx];
  output_data[id] = input_data[input_index];
}

Status SliceImpl(const size_t element_size,
               const int32_t dimension_count,
               const int64_t* starts,
               const int64_t* steps,
               const int64_t* input_strides,
               const fast_divmod* output_div_strides,
               const void* input_data,
               void* output_data,
               const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  switch (element_size) {
    case sizeof(int8_t):
      _SliceKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_div_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    case sizeof(int16_t):
      _SliceKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_div_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    case sizeof(int32_t):
      _SliceKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_div_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    case sizeof(int64_t):
      _SliceKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_div_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Slice operator");
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
