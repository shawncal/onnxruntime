#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "transpose_impl.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _TransposeKernel(size_t shape_rank, const int64_t* input_strides, const size_t* perm,
                                 const T* input_data, const fast_divmod* fdm_output_strides, T* output_data, size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

  for (int dim = 0; dim < shape_rank; ++dim) {
    int out_coord, r;
    fdm_output_strides[dim].divmod(output_index, out_coord, r);
    output_index = r;
    input_index += input_strides[perm[dim]] * out_coord;
  }
  output_data[id] = input_data[input_index];
}

template <typename T>
void TransposeImpl(size_t shape_rank, const int64_t* input_strides, const size_t* perm, const T* input_data,
                   const fast_divmod* fdm_output_strides, T* output_data, size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  _TransposeKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      shape_rank, input_strides, perm, input_data,
      fdm_output_strides, output_data, N);
}

#define SPECIALIZED_IMPL(T)                                                                                  \
  template void TransposeImpl<T>(size_t shape_rank, const int64_t* input_strides, const size_t* perm,        \
                                 const T* input_data, const fast_divmod* fdm_output_strides, T* output_data, \
                                 size_t N);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(half)

}  // namespace cuda
}  // namespace onnxruntime
